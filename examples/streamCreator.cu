#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>

using namespace std;
hipStream_t createStreamWithFlags(){
	hipStream_t stream;
	hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
	return stream;
}
