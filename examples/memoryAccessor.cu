#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>

using namespace std;
void cuSetDeviceFlags(){
	hipSetDeviceFlags(hipDeviceMapHost);
}
void cuMallocManaged(void** h_img, int r, int c, int channel){

	hipMallocManaged(h_img,sizeof(unsigned char)*r*c * channel);

}

void cuMalloc(void** h_img, int r, int c){
	hipMalloc(h_img, sizeof(float)*r*c);
}

void cuDeviceSynchronize(){
	hipDeviceSynchronize();
}

void cuFree(void* mem){
	hipFree(mem);
}
